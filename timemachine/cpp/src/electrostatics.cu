#include <chrono>
#include <iostream>
#include <vector>
#include <complex>
#include "electrostatics.hpp"
#include "gpu_utils.cuh"

#include "k_electrostatics.cuh"
#include "k_electrostatics_jvp.cuh"

namespace timemachine {

template <typename RealType>
Electrostatics<RealType>::Electrostatics(
    const std::vector<double> &charge_params, // [N]
    const std::vector<int> &exclusion_idxs, // [E,2]
    const std::vector<double> &charge_scales, // [E]
    const std::vector<int> &lambda_plane_idxs, // [N]
    const std::vector<int> &lambda_offset_idxs, // [N]
    double cutoff
) :  N_(charge_params.size()),
    cutoff_(cutoff),
    E_(exclusion_idxs.size()/2),
    nblist_(charge_params.size(), 3) {

    if(lambda_plane_idxs.size() != N_) {
        throw std::runtime_error("lambda plane idxs need to have size N");
    }

    if(lambda_offset_idxs.size() != N_) {
        throw std::runtime_error("lambda offset idxs need to have size N");
    }

    if(charge_scales.size()*2 != exclusion_idxs.size()) {
        throw std::runtime_error("charge scale idxs size not half of exclusion size!");
    }

    gpuErrchk(hipMalloc(&d_lambda_plane_idxs_, N_*sizeof(*d_lambda_plane_idxs_)));
    gpuErrchk(hipMemcpy(d_lambda_plane_idxs_, &lambda_plane_idxs[0], N_*sizeof(*d_lambda_plane_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_lambda_offset_idxs_, N_*sizeof(*d_lambda_offset_idxs_)));
    gpuErrchk(hipMemcpy(d_lambda_offset_idxs_, &lambda_offset_idxs[0], N_*sizeof(*d_lambda_offset_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_exclusion_idxs_, E_*2*sizeof(*d_exclusion_idxs_)));
    gpuErrchk(hipMemcpy(d_exclusion_idxs_, &exclusion_idxs[0], E_*2*sizeof(*d_exclusion_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_charge_scales_, E_*sizeof(*d_charge_scales_)));
    gpuErrchk(hipMemcpy(d_charge_scales_, &charge_scales[0], E_*sizeof(*d_charge_scales_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_charge_params_, N_*sizeof(*d_charge_params_)));
    gpuErrchk(hipMemcpy(d_charge_params_, &charge_params[0], N_*sizeof(*d_charge_params_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_du_dcharge_primals_, N_*sizeof(*d_du_dcharge_primals_)));
    gpuErrchk(hipMemset(d_du_dcharge_primals_, 0, N_*sizeof(*d_du_dcharge_primals_)));

    gpuErrchk(hipMalloc(&d_du_dcharge_tangents_, N_*sizeof(*d_du_dcharge_tangents_)));
    gpuErrchk(hipMemset(d_du_dcharge_tangents_, 0, N_*sizeof(*d_du_dcharge_tangents_)));


};

template <typename RealType>
Electrostatics<RealType>::~Electrostatics() {
    gpuErrchk(hipFree(d_charge_params_));
    gpuErrchk(hipFree(d_exclusion_idxs_));
    gpuErrchk(hipFree(d_charge_scales_));
    gpuErrchk(hipFree(d_lambda_plane_idxs_));
    gpuErrchk(hipFree(d_lambda_offset_idxs_));

    gpuErrchk(hipFree(d_du_dcharge_primals_));
    gpuErrchk(hipFree(d_du_dcharge_tangents_));
};


template <typename RealType>
void Electrostatics<RealType>::get_du_dcharge_primals(double *buf) {
    gpuErrchk(hipMemcpy(buf, d_du_dcharge_primals_, N_*sizeof(*d_du_dcharge_primals_), hipMemcpyDeviceToHost));
}

template <typename RealType>
void Electrostatics<RealType>::get_du_dcharge_tangents(double *buf) {
    gpuErrchk(hipMemcpy(buf, d_du_dcharge_tangents_, N_*sizeof(*d_du_dcharge_tangents_), hipMemcpyDeviceToHost));
}

template <typename RealType>
void Electrostatics<RealType>::execute_lambda_inference_device(
    const int N,
    const double *d_coords_primals,
    const double lambda_primal,
    unsigned long long *d_out_coords_primals,
    double *d_out_lambda_primals,
    double *d_out_energy_primal,
    hipStream_t stream) {

    if(N != N_) {
        throw std::runtime_error("N != N_");
    }

    const int tpb = 32;
    const int B = (N_+tpb-1)/tpb;
    const int D = 3;

    // its safe for us to build a neighborlist in a lower dimension.
    nblist_.compute_block_bounds(N_, D, d_coords_primals, stream);

    gpuErrchk(hipPeekAtLastError());

    dim3 dimGrid(B, B, 1); // x, y, z dims
    dim3 dimGridExclusions((E_+tpb-1)/tpb, 1, 1);

    auto start = std::chrono::high_resolution_clock::now();

    // these can be ran in two streams later on
    k_electrostatics_inference<RealType><<<dimGrid, tpb, 0, stream>>>(
        N_,
        d_coords_primals,

        lambda_primal,
        d_lambda_plane_idxs_,
        d_lambda_offset_idxs_,
        d_charge_params_,
        cutoff_,
        nblist_.get_block_bounds_ctr(),
        nblist_.get_block_bounds_ext(),
        d_out_coords_primals,
        d_out_lambda_primals,
        d_out_energy_primal
    );

    // hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    if(E_ > 0) {
        k_electrostatics_exclusion_inference<RealType><<<dimGridExclusions, tpb, 0, stream>>>(
            E_,
            d_coords_primals,
            lambda_primal,
            d_lambda_plane_idxs_,
            d_lambda_offset_idxs_,
            d_exclusion_idxs_,
            d_charge_scales_,
            d_charge_params_,
            cutoff_,
            d_out_coords_primals,
            d_out_lambda_primals,
            d_out_energy_primal
        );
        // hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }
}


template <typename RealType>
void Electrostatics<RealType>::execute_lambda_jvp_device(
    const int N,
    const double *d_coords_primals,
    const double *d_coords_tangents,
    const double lambda_primal,
    const double lambda_tangent,
    double *d_out_coords_primals,
    double *d_out_coords_tangents,
    hipStream_t stream) {

    if(N != N_) {
        throw std::runtime_error("N != N_");
    }

    const int tpb = 32;
    const int B = (N_+tpb-1)/tpb;
    const int D = 3;

    nblist_.compute_block_bounds(N_, D, d_coords_primals, stream);

    gpuErrchk(hipPeekAtLastError());

    dim3 dimGrid(B, B, 1); // x, y, z dims
    dim3 dimGridExclusions((E_+tpb-1)/tpb, 1, 1);

    auto start = std::chrono::high_resolution_clock::now();

    k_electrostatics_jvp<RealType><<<dimGrid, tpb, 0, stream>>>(
        N_,
        d_coords_primals,
        d_coords_tangents,
        lambda_primal,
        lambda_tangent,
        d_lambda_plane_idxs_,
        d_lambda_offset_idxs_,
        d_charge_params_,
        cutoff_,
        nblist_.get_block_bounds_ctr(),
        nblist_.get_block_bounds_ext(),
        d_out_coords_primals,
        d_out_coords_tangents,
        d_du_dcharge_primals_,
        d_du_dcharge_tangents_
    );

    // hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    if(E_ > 0) {
        k_electrostatics_exclusion_jvp<RealType><<<dimGridExclusions, tpb, 0, stream>>>(
            E_,
            d_coords_primals,
            d_coords_tangents,
            lambda_primal,
            lambda_tangent,
            d_lambda_plane_idxs_,
            d_lambda_offset_idxs_,
            d_exclusion_idxs_,
            d_charge_scales_,
            d_charge_params_,
            cutoff_,
            d_out_coords_primals,
            d_out_coords_tangents,
            d_du_dcharge_primals_,
            d_du_dcharge_tangents_
        );            

        // hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }



};

template class Electrostatics<double>;
template class Electrostatics<float>;

} // namespace timemachine