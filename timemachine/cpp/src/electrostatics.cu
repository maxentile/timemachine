#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <vector>
#include <complex>
#include <sstream>
#include <numeric>
#include <algorithm>

#include "hilbert.h"
#include "electrostatics.hpp"
#include "gpu_utils.cuh"

#include "k_electrostatics.cuh"
#include "k_electrostatics_jvp.cuh"

namespace timemachine {

template <typename RealType>
Electrostatics<RealType>::Electrostatics(
    const std::vector<int> &exclusion_idxs, // [E,2]
    const std::vector<double> &charge_scales, // [E]
    const std::vector<int> &lambda_offset_idxs, // [N]
    double beta,
    double cutoff
) :  N_(lambda_offset_idxs.size()),
    beta_(beta),
    cutoff_(cutoff),
    E_(exclusion_idxs.size()/2),
    nblist_(lambda_offset_idxs.size(), 3),
    d_perm_(nullptr) {

    if(lambda_offset_idxs.size() != N_) {
        throw std::runtime_error("lambda offset idxs need to have size N");
    }

    if(charge_scales.size()*2 != exclusion_idxs.size()) {
        throw std::runtime_error("charge scale idxs size not half of exclusion size!");
    }

    gpuErrchk(hipMalloc(&d_lambda_offset_idxs_, N_*sizeof(*d_lambda_offset_idxs_)));
    gpuErrchk(hipMemcpy(d_lambda_offset_idxs_, &lambda_offset_idxs[0], N_*sizeof(*d_lambda_offset_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_exclusion_idxs_, E_*2*sizeof(*d_exclusion_idxs_)));
    gpuErrchk(hipMemcpy(d_exclusion_idxs_, &exclusion_idxs[0], E_*2*sizeof(*d_exclusion_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_charge_scales_, E_*sizeof(*d_charge_scales_)));
    gpuErrchk(hipMemcpy(d_charge_scales_, &charge_scales[0], E_*sizeof(*d_charge_scales_), hipMemcpyHostToDevice));


};

template <typename RealType>
Electrostatics<RealType>::~Electrostatics() {
    gpuErrchk(hipFree(d_exclusion_idxs_));
    gpuErrchk(hipFree(d_charge_scales_));
    gpuErrchk(hipFree(d_lambda_offset_idxs_));
    gpuErrchk(hipFree(d_perm_));
};

struct Vec3 {

    double x, y, z;

    Vec3(double x, double y, double z) : x(x), y(y), z(z) {};

    Vec3 operator*(double a) {
        return Vec3(x*a, y*a, z*a);
    }


};

template <typename RealType>
void Electrostatics<RealType>::execute_device(
    const int N,
    const int P,
    const double *d_x,
    const double *d_p,
    const double *d_box,
    const double lambda,
    unsigned long long *d_du_dx,
    double *d_du_dp,
    double *d_du_dl,
    double *d_u,
    hipStream_t stream) {

    if(N != N_) {
        std::ostringstream err_msg;
        err_msg << "N != N_ " << N << " " << N_;
        throw std::runtime_error(err_msg.str());
    }

    const int tpb = 32;
    const int B = (N_+tpb-1)/tpb;
    const int D = 3;



    // add conditional for flipping a random number.
    bool sort = 0;
    if(d_perm_ == nullptr) {
        gpuErrchk(hipMalloc(&d_perm_, N_*sizeof(*d_perm_)));
        sort = true;
    }

    // we will add a predicate that re-shuffles once every 30 steps or so on average

    auto start_sort = std::chrono::high_resolution_clock::now();

    // can probably be turned into a float for speed, since the nblist is approximate anyways

    if(sort) {
        // tbd switch with asyncversions
        std::vector<double> box(9);
        gpuErrchk(hipMemcpy(&box[0], d_box, 9*sizeof(double), hipMemcpyDeviceToHost));

        // 1. copy over coordinates
        std::vector<double> coords(N_*3);
        gpuErrchk(hipMemcpy(&coords[0], d_x, N_*3*sizeof(double), hipMemcpyDeviceToHost));
     
        std::vector<double> centered_coords(N_*3);

        double bx = box[0*3+0];
        double by = box[1*3+1];
        double bz = box[2*3+2];

        // 2. apply periodic centering
        for(int i=0; i < N_; i++) {

            double x = coords[i*3+0];
            double y = coords[i*3+1];
            double z = coords[i*3+2];

            // only if periodic
            x -= bx*floor(x/bx);
            y -= by*floor(y/by);
            z -= bz*floor(z/bz);

            centered_coords[i*3+0] = x;
            centered_coords[i*3+1] = y;
            centered_coords[i*3+2] = z;
        }
        
        // 3. build the hilbert curve
        // if periodic
        // double minx = 0.0;
        // double miny = 0.0;
        // double minz = 0.0;
        // double maxx = bx;
        // double maxy = by;
        // double maxz = bz;

        // always use this to generate the bounding box
        double minx = centered_coords[0*3+0], maxx = centered_coords[0*3+0];
        double miny = centered_coords[0*3+1], maxy = centered_coords[0*3+1];
        double minz = centered_coords[0*3+2], maxz = centered_coords[0*3+2];
        for (int i = 1; i < N_; i++) {
            // const Real4& pos = oldPosq[i];
            minx = min(minx, centered_coords[i*3+0]);
            maxx = max(maxx, centered_coords[i*3+0]);
            miny = min(miny, centered_coords[i*3+1]);
            maxy = max(maxy, centered_coords[i*3+1]);
            minz = min(minz, centered_coords[i*3+2]);
            maxz = max(maxz, centered_coords[i*3+2]);
        }

        double binWidth = max(max(maxx-minx, maxy-miny), maxz-minz)/255.0;
        double invBinWidth = 1.0/binWidth;
        std::vector<std::pair<int, int> > molBins(N_);

        for(int i = 0; i < N_; i++) {
            int x = (centered_coords[i*3+0]-minx)*invBinWidth;
            int y = (centered_coords[i*3+1]-miny)*invBinWidth;
            int z = (centered_coords[i*3+2]-minz)*invBinWidth;

            bitmask_t hilbert_coords[3];
            hilbert_coords[0] = x;
            hilbert_coords[1] = y;
            hilbert_coords[2] = z;
            int bin = (int) hilbert_c2i(3, 8, hilbert_coords);

            molBins[i] = std::pair<int, int>(bin, i);
        }
        std::sort(molBins.begin(), molBins.end());
        // 4. generate a new ordering
        std::vector<int> perm(N_);
        for(int i=0; i < N_; i++) {
            perm[i] = molBins[i].second;
        }
        gpuErrchk(hipMemcpy(d_perm_, &perm[0], N*sizeof(*d_perm_), hipMemcpyHostToDevice));
    }

    auto end_sort = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_sort - start_sort).count();
    std::cout << duration << "us to re-sort" << std::endl;;

    // its safe for us to build a neighborlist in a lower dimension.
    nblist_.compute_block_bounds(
        N_,
        D,
        d_x,
        d_box,
        d_perm_,
        stream
    );

    std::vector<double> bb_ctr(B*3);
    std::vector<double> bb_ext(B*3);

    gpuErrchk(hipMemcpy(&bb_ctr[0], nblist_.get_block_bounds_ctr(), B*3*sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(&bb_ext[0], nblist_.get_block_bounds_ext(), B*3*sizeof(double), hipMemcpyDeviceToHost));

    std::vector<int> tiles_x;
    std::vector<int> tiles_y;

    std::vector<double> box(9);
    gpuErrchk(hipMemcpy(&box[0], d_box, 9*sizeof(double), hipMemcpyDeviceToHost));

    double bx[3] = {box[0*3+0], box[1*3+1], box[2*3+2]};

    std::vector<double> bv;


    for(int x=0; x < B; x++) {

        double vol = 1;

        for(int d=0; d < 3; d++) {   
            double block_row_ext = bb_ext[x*3+d];
            vol *= 2*block_row_ext*2*block_row_ext;
        }
        bv.push_back(vol);

    }

    double box_sum = std::accumulate(bv.begin(), bv.end(), 0.0);
    double box_mean = box_sum / bv.size();

    double box_sq_sum = std::inner_product(bv.begin(), bv.end(), bv.begin(), 0.0);
    double box_stdev = std::sqrt(box_sq_sum / bv.size() - box_mean * box_mean);

    std::cout << "box mean " << box_mean << " std " << box_stdev << std::endl;

    // throw std::runtime_error("debug");

    // check bounding box deltas
    for(int x=0; x < B; x++) {

        for(int y=0; y < B; y++) {

            if(y > x) {
                continue;
            }

            double block_d2ij = 0;
 
            for(int d=0; d < 3; d++) {
                double block_row_ctr = bb_ctr[x*3+d];
                double block_row_ext = bb_ext[x*3+d];
                double block_col_ctr = bb_ctr[y*3+d];
                double block_col_ext = bb_ext[y*3+d];

                double dx = block_row_ctr - block_col_ctr;
                dx -= bx[d]*floor(dx/bx[d]+static_cast<double>(0.5));
                dx = max(static_cast<double>(0.0), fabs(dx) - (block_row_ext + block_col_ext));
                block_d2ij += dx*dx;                
            }

            if(block_d2ij < cutoff_*cutoff_) {
                tiles_x.push_back(x);
                tiles_y.push_back(y);
            } else{
                // std::cout << "skipping: " << x << " " << y << std::endl;
            }

        }
    }

    int *d_tiles_x_ = gpuErrchkCudaMallocAndCopy(&tiles_x[0], tiles_x.size());
    int *d_tiles_y_ = gpuErrchkCudaMallocAndCopy(&tiles_y[0], tiles_y.size());


    std::cout << "num_tiles: " << tiles_x.size() << " out of " << (N_/32)*(N_/32) << std::endl;


    gpuErrchk(hipPeekAtLastError());

    // remove me later
    hipDeviceSynchronize();

    // const int TILES = (tiles_x.size()+tpb-1)/tpb;
    const int TILES = tiles_x.size();

    // dim3 dimGrid(B, B, 1); // x, y, z dims
    dim3 dimGrid(TILES, 1, 1); // x, y, z dims
    dim3 dimGridExclusions((E_+tpb-1)/tpb, 1, 1);

    auto start = std::chrono::high_resolution_clock::now();


    // these can be ran in two streams later on
    int *total_ixns;
    gpuErrchk(hipMallocManaged(&total_ixns, 1*sizeof(int)));

    int *total_empty_tiles;
    gpuErrchk(hipMallocManaged(&total_empty_tiles, 1*sizeof(int)));


    k_electrostatics<RealType><<<dimGrid, tpb, 0, stream>>>(
        N_,
        d_x,
        d_p,
        d_box,
        lambda,
        d_lambda_offset_idxs_,
        beta_,
        cutoff_,
        // nblist_.get_block_bounds_ctr(),
        // nblist_.get_block_bounds_ext(),
        d_tiles_x_,
        d_tiles_y_,
        d_perm_,
        d_du_dx,
        d_du_dp,
        d_du_dl,
        d_u,
        total_ixns,
        total_empty_tiles
    );

    hipDeviceSynchronize();

    std::cout << "total ixns: " << *total_ixns << "/" << TILES*(32*32) << std::endl;

    std::cout << "total empty tiles: " << *total_empty_tiles << "/" << tiles_x.size() << std::endl;

    // hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    if(E_ > 0) {
        // k_electrostatics_exclusion_inference<RealType><<<dimGridExclusions, tpb, 0, stream>>>(
        //     E_,
        //     d_x,
        //     d_p,
        //     d_box,
        //     lambda,
        //     d_lambda_offset_idxs_,
        //     d_exclusion_idxs_,
        //     d_charge_scales_,
        //     beta_,
        //     cutoff_,
        //     d_du_dx,
        //     d_du_dp,
        //     d_du_dl,
        //     d_u
        // );
        // hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }
}

template class Electrostatics<double>;
template class Electrostatics<float>;

} // namespace timemachine