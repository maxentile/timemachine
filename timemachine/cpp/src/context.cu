#include "context.hpp"
#include "gpu_utils.cuh"
#include <iostream>

namespace timemachine {

Context::Context(
    int N,
    const double *x_0,
    const double *v_0,
    const double *box_0,
    // double lambda,
    Integrator* intg,
    std::vector<BoundPotential *> bps) :
    // std::vector<Observable *> obs) : 
    N_(N),
    intg_(intg),
    bps_(bps),
    // observables_(obs),
    // lambda_(lambda),
    step_(0) {

    d_x_t_ = gpuErrchkCudaMallocAndCopy(x_0, N*3);
    d_v_t_ = gpuErrchkCudaMallocAndCopy(v_0, N*3);
    d_box_t_ = gpuErrchkCudaMallocAndCopy(box_0, 3*3);

    gpuErrchk(hipMalloc(&d_du_dx_t_, N*3*sizeof(*d_du_dx_t_)));
    gpuErrchk(hipMalloc(&d_u_t_, 1*sizeof(*d_u_t_)));

};

Context::~Context() {
    gpuErrchk(hipFree(d_x_t_));
    gpuErrchk(hipFree(d_v_t_));
    gpuErrchk(hipFree(d_box_t_));
    gpuErrchk(hipFree(d_u_t_));
    gpuErrchk(hipFree(d_du_dx_t_));
};

void Context::add_observable(Observable *obs) {
    this->observables_.push_back(obs);
}

void Context::step(double lambda) {

    // the observables decide on whether or not to act on given
    // data (cheap pointers in any case)

    for(int i=0; i < observables_.size(); i++) {
        observables_[i]->observe(
            step_,
            N_,
            d_x_t_,
            d_box_t_,
            lambda
        );
    }

    gpuErrchk(hipMemset(d_du_dx_t_, 0, N_*3*sizeof(*d_du_dx_t_)));
    gpuErrchk(hipMemset(d_u_t_, 0, 1*sizeof(*d_du_dx_t_)));

    for(int i=0; i < bps_.size(); i++) {
        // std::cout << i << std::endl;
        bps_[i]->execute_device(
            N_,
            d_x_t_,
            d_box_t_,
            lambda,
            d_du_dx_t_,
            nullptr,
            nullptr,
            d_u_t_,
            static_cast<hipStream_t>(0) // TBD: parallelize me!
        );
    }

    intg_->step_fwd(
        d_x_t_,
        d_v_t_,
        d_du_dx_t_,
        d_box_t_
    );

    hipDeviceSynchronize();

    step_ += 1;

};


int Context::num_atoms() const {
    return N_;
}

double Context::get_u_t() const {
    double u;
    gpuErrchk(hipMemcpy(&u, d_u_t_, 1*sizeof(*d_u_t_), hipMemcpyDeviceToHost));
    return u;
}

void Context::get_du_dx_t(unsigned long long *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_du_dx_t_, N_*3*sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_x_t(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_x_t_, N_*3*sizeof(*out_buffer), hipMemcpyDeviceToHost));
}

void Context::get_v_t(double *out_buffer) const {
    gpuErrchk(hipMemcpy(out_buffer, d_v_t_, N_*3*sizeof(*out_buffer), hipMemcpyDeviceToHost));
}


// void Context::compute(unsigned int flags) {

//     double *u = (flags & ComputeFlags::u) ? d_u_t_ : nullptr;
//     unsigned long long *du_dx = (flags & ComputeFlags::du_dx) ? d_du_dx_t_ : nullptr;
//     double *du_dl = (flags & ComputeFlags::du_dl) ? d_du_dl_t_ : nullptr;

//     for(int i=0; i < potentials_.size(); i++) {

//         DualParams *dp = dual_params_[i];

//         // note that dp->d_du_dp itself may be null if the end-user
//         // does not care about du_dp.
//         double *du_dp = (flags & ComputeFlags::du_dp) ? dp->d_du_dp : nullptr;

//         potentials_[i]->execute_device(
//             N_,
//             dp->size(),
//             d_x_t_,
//             dp->d_p,
//             d_box_t_,
//             lambda_,
//             du_dx,
//             du_dp,
//             du_dl,
//             u,
//             static_cast<hipStream_t>(0)
//         );

//     }

// };


}