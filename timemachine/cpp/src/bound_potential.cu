#include "bound_potential.hpp"
#include "gpu_utils.cuh"

namespace timemachine {

BoundPotential::BoundPotential(
    std::vector<int> shape,
    double *h_p
) : shape(shape) {

    int P = this->size();

    gpuErrchk(hipMalloc(&d_p, P*sizeof(*d_p)));
    gpuErrchk(hipMemcpy(d_p, h_p, P*sizeof(*d_p), hipMemcpyHostToDevice));

}

BoundPotential::~BoundPotential() {
    // only free the d_ps, but not the pure potentials themselves
    gpuErrchk(hipFree(d_p));
}

int BoundPotential::size() const {
    if(shape.size() == 0) {
        return 0;
    }
    int total = 1;
    for(auto s : shape) {
        total *= s;
    }
    return total;
}

}
